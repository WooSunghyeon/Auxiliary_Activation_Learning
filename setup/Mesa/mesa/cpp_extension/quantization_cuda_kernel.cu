#include "hip/hip_runtime.h"
// Copyright (c) 2021-present, Zhuang AI Group.
// All rights reserved.

#include <torch/extension.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_Y_DIM_MAX ((((int64_t)(1)) << 16) - 1)
#define fmax(a, b) ((a) > (b) ? (a) : (b))
#include <c10/cuda/CUDAGuard.h>


using torch::IntArrayRef;
using torch::Tensor;

// Pack float16/32 data into int8 bit stream
template <typename scalar_t, bool boundary_check>
__global__ void pack_single_precision_kernel(int32_t bits,
                                             const scalar_t *__restrict__ data,
                                             const scalar_t *__restrict__ scale,
                                             const scalar_t *__restrict__ shift,
                                             int8_t *__restrict__ packed,
                                             std::pair<uint64_t, uint64_t> seeds)
{
  const int64_t global_thread_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t group_id = blockIdx.y;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, global_thread_id, seeds.second, &state);
  const float noise = hiprand_uniform(&state);

  uint8_t local_packed = 0;
  const int32_t val = __float2int_rn(fmax((data[id] - shift[group_id]) * scale[group_id] + noise - 0.5, 0.0f));
  local_packed |= val;
  packed[global_thread_id] = local_packed;
}

// Pack float16/32 data into int8 bit stream
Tensor pack_single_precision_cuda(Tensor data,
                                  Tensor scale,
                                  Tensor shift,
                                  int bits,
                                  bool stochastic)
{
  at::cuda::CUDAGuard device_guard(data.device());
  int64_t num_groups = data.size(0);
  int64_t group_size = data.size(1);

  // Compute total bits
  TORCH_CHECK(8 % bits == 0);
  int64_t total_bits = (int64_t)bits * (num_groups * group_size);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(data.device());
  Tensor packed = torch::empty({
                                   (total_bits + 8) / 8,
                               },
                               options);

  int threads = 1024;
  while (group_size % threads != 0 && threads > 1)
  {
    threads /= 2;
  }
  int blockdim_x = group_size / threads;
  dim3 block_dim(blockdim_x, num_groups, 1);
  dim3 thread_dim(threads, 1, 1);

  // Random number generator
  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(threads);
  }

  //   TORCH_CHECK(stochastic);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_single_precision", ([&]
                                                                                    { pack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
                                                                                          bits,
                                                                                          data.data_ptr<scalar_t>(),
                                                                                          scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
                                                                                          packed.data_ptr<int8_t>(),
                                                                                          rng_engine_inputs); }));
  return packed;
}

// Unpack int32 bit stream to float16/32 data
template <typename scalar_t, bool boundary_check>
__global__ void unpack_single_precision_kernel(int32_t bits,
                                               const int8_t *__restrict__ data,
                                               const scalar_t *__restrict__ scale,
                                               const scalar_t *__restrict__ shift,
                                               scalar_t *__restrict__ unpacked)
{
  const int64_t global_thread_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  uint8_t local_packed = data[global_thread_id];
  int mask = ((1 << bits) - 1);
  const int val = local_packed & mask;
  const int64_t id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t group_id = blockIdx.y;
  unpacked[id] = ((scalar_t)val) / scale[group_id] + shift[group_id];
}

// Unpack int32 bit stream to float16/32 data
Tensor unpack_single_precision_cuda(Tensor data,
                                    int bits,
                                    Tensor scale,
                                    Tensor shift,
                                    int64_t num_groups,
                                    int64_t group_size)
{
  auto options = torch::TensorOptions().dtype(scale.dtype()).device(data.device());
  Tensor unpacked = torch::empty({num_groups, group_size}, options);
  int threads = 1024;
  while (group_size % threads != 0 && threads > 1)
  {
    threads /= 2;
  }
  int blockdim_x = group_size / threads;
  dim3 block_dim(blockdim_x, num_groups, 1);
  dim3 thread_dim(threads, 1, 1);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_single_precision", ([&]
                                                                                       { unpack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
                                                                                             bits,
                                                                                             data.data_ptr<int8_t>(),
                                                                                             scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
                                                                                             unpacked.data_ptr<scalar_t>()); }));
  return unpacked;
}
